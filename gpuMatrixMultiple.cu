#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define SIZE        2
#define BLOCK_SIZE  1

void showMatrix(int *matrix);
__global__ void matrixMultiple(int *matrixA, int *matrixB, int *matrixC);


int main(int argc, char* argv[])
{
    const size_t matrixMemSize = sizeof(int) * SIZE * SIZE;

    // ホスト側のメモリ領域確保
    int *hostA, *hostB, *hostC;
    hostA = (int *) malloc( matrixMemSize );
    hostB = (int *) malloc( matrixMemSize );
    hostC = (int *) malloc( matrixMemSize );

    // 乱数系列の初期化
    srandom( (unsigned) time(NULL) );
    // 初期化処理
    for (int y = 0; y < SIZE; y++) {
        for (int x = 0; x < SIZE; x++) {
            hostA[y * SIZE + x] = random() % 50;
            hostB[y * SIZE + x] = random() % 50;
            hostC[y * SIZE + x] = 0;
        }
    }

    // デバイス側のメモリ領域確保 & データ転送
    int *deviceA, *deviceB, *deviceC;
    hipMalloc( (void **)&deviceA, matrixMemSize );
    hipMalloc( (void **)&deviceB, matrixMemSize );
    hipMalloc( (void **)&deviceC, matrixMemSize );
    hipMemcpy( deviceA, hostA, matrixMemSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceB, hostB, matrixMemSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceC, hostC, matrixMemSize, hipMemcpyHostToDevice );


    // グリッド & ブロックサイズの設定
    dim3 grid(SIZE/BLOCK_SIZE, SIZE/BLOCK_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // 時間計測開始
    hipEvent_t  start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    // 行列積を計算
    matrixMultiple<<< grid, block >>>( deviceA, deviceB, deviceC );
    hipDeviceSynchronize();

    // 時間計測終了
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    // データ転送: device -> host
    hipMemcpy( hostC, deviceC, matrixMemSize, hipMemcpyDeviceToHost );

    // 結果表示
    // puts("matrixA =");
    // showMatrix( hostA );
    // puts("matrixB =");
    // showMatrix( hostB );
    // puts("matrixC =");
    // showMatrix( hostC );

    // 計測結果表示
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf("elapsed time: %f ms\n", elapsedTime);

    hipEventDestroy( start );
    hipEventDestroy( stop );


    // デバイス側のメモリ領域解放
    hipFree( deviceA );
    hipFree( deviceB );
    hipFree( deviceC );

    // ホスト側のメモリ領域解放
    free( hostA );
    free( hostB );
    free( hostC );

    return 0;
}


void showMatrix(int *matrix)
{
    for (int y = 0; y < SIZE; y++) {
        for (int x = 0; x < SIZE; x++) {
            printf("%5d ", matrix[y * SIZE + x]);
        }
        puts("");
    }
}


__global__ 
void matrixMultiple(int *matrixA, int *matrixB, int *matrixC)
{
    unsigned int   x = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int   y = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int idx = (y * SIZE) + x;

    int value = 0;

#ifdef _USE_SHARED_MEM
    // SharedMemory を使う場合:
    unsigned int tx = threadIdx.x,  bx = blockIdx.x;
    unsigned int ty = threadIdx.y,  by = blockIdx.y;

    __shared__ int sharedMatA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sharedMatB[BLOCK_SIZE][BLOCK_SIZE];

    for (int i = 0; i < SIZE/BLOCK_SIZE; i++) {

        unsigned int px = (SIZE * BLOCK_SIZE * by) + (i * BLOCK_SIZE);
        unsigned int py = (BLOCK_SIZE * bx) + SIZE * (i * BLOCK_SIZE);

        sharedMatA[ty][tx] = matrixA[px + (SIZE * ty + tx)];
        sharedMatB[ty][tx] = matrixB[py + (SIZE * ty + tx)];
        __syncthreads();

        for (int j = 0; j < BLOCK_SIZE; j++) {
            value += (sharedMatA[ty][j] * sharedMatB[j][tx]);
        }
        __syncthreads();
    }
#else
    // SharedMemory を使わない場合:
    for (int i = 0; i < SIZE; i++) {
        value += matrixA[(y * SIZE) + i] * matrixB[(i * SIZE) + x];
        __syncthreads();
    }
#endif
    matrixC[idx] = value;
}
