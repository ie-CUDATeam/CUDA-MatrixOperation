#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define SIZE       16
#define BLOCK_SIZE  1

void showMatrix(int *matrix);
__global__ void matrixMultiple(int *matrixA, int *matrixB, int *matrixC);


int main(int argc, char* argv[])
{
    const size_t matrixSize = sizeof(int) * SIZE * SIZE;

    // ホスト側のメモリ領域確保
    int *hostA, *hostB, *hostC;
    hostA = (int *) malloc( matrixSize );
    hostB = (int *) malloc( matrixSize );
    hostC = (int *) malloc( matrixSize );

    // 乱数系列の初期化
    srandom( (unsigned) time(NULL) );
    // 初期化処理
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            hostA[i * SIZE + j] = random() % 50;
            hostB[i * SIZE + j] = random() % 50;
            hostC[i * SIZE + j] = 0;
        }
    }

    // デバイス側のメモリ領域確保 & データ転送
    int *deviceA, *deviceB, *deviceC;
    hipMalloc( (void **)&deviceA, matrixSize );
    hipMalloc( (void **)&deviceB, matrixSize );
    hipMalloc( (void **)&deviceC, matrixSize );
    hipMemcpy( deviceA, hostA, matrixSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceB, hostB, matrixSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceC, hostC, matrixSize, hipMemcpyHostToDevice );


    // グリッド & ブロックサイズの設定
    dim3 grid(BLOCK_SIZE, BLOCK_SIZE);
    dim3 block(SIZE/BLOCK_SIZE, SIZE/BLOCK_SIZE);
    // 行列積を計算
    matrixMultiple<<< grid, block >>>( deviceA, deviceB, deviceC );
    // データ転送: device -> host
    hipMemcpy( hostC, deviceC, matrixSize, hipMemcpyDeviceToHost );


    // 結果表示
    puts("matrixA =");
    showMatrix( hostA );
    puts("matrixB =");
    showMatrix( hostB );
    puts("matrixC =");
    showMatrix( hostC );


    // デバイス側のメモリ領域解放
    hipFree( deviceA );
    hipFree( deviceB );
    hipFree( deviceC );

    // ホスト側のメモリ領域解放
    free( hostA );
    free( hostB );
    free( hostC );

    return 0;
}


void showMatrix(int *matrix)
{
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            printf("%5d ", matrix[i * SIZE + j]);
        }
        puts("");
    }
}


__global__ 
void matrixMultiple(int *matrixA, int *matrixB, int *matrixC)
{
    unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    int value = 0;
    for (int i = 0; i < SIZE; i++) {
        value += matrixA[(y * SIZE) + i] * matrixB[(i * SIZE) + x];
        __syncthreads();
    }
    matrixC[(y * SIZE) + x] = value;
}
