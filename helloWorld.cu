#include <iostream>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


int main(int argc, char *argv[])
{
    using namespace std;

    int devId = findCudaDevice(argc, (const char **)argv);
    cout << "devId = " << devId << endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devId);
    cout << "multiProcessorCount = " << prop.multiProcessorCount << endl;
    cout << "deviceOverlap = " << prop.deviceOverlap << endl;

    return 0;
}
