#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define SIZE       32
#define BLOCK_SIZE  1

void showMatrix(int *matrix);
__global__ void matrixSum(int *matrixA, int *matrixB, int *matrixC);


int main(int argc, char* argv[])
{
    // 時間計測開始
    hipEvent_t  start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );


    const size_t matrixSize = sizeof(int) * SIZE * SIZE;

    // ホスト側のメモリ領域確保
    int *hostA, *hostB, *hostC;
    hostA = (int *) malloc( matrixSize );
    hostB = (int *) malloc( matrixSize );
    hostC = (int *) malloc( matrixSize );

    // 乱数系列の初期化
    srandom( (unsigned) time(NULL) );
    // 初期化処理
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            hostA[i * SIZE + j] = random() % 50;
            hostB[i * SIZE + j] = random() % 50;
            hostC[i * SIZE + j] = 0;
        }
    }

    // デバイス側のメモリ領域確保 & データ転送
    int *deviceA, *deviceB, *deviceC;
    hipMalloc( (void **)&deviceA, matrixSize );
    hipMalloc( (void **)&deviceB, matrixSize );
    hipMalloc( (void **)&deviceC, matrixSize );
    hipMemcpy( deviceA, hostA, matrixSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceB, hostB, matrixSize, hipMemcpyHostToDevice );
    hipMemcpy( deviceC, hostC, matrixSize, hipMemcpyHostToDevice );


    // グリッド & ブロックサイズの設定
    dim3 grid(BLOCK_SIZE, BLOCK_SIZE);
    dim3 block(SIZE/BLOCK_SIZE, SIZE/BLOCK_SIZE);
    // 行列和を計算
    matrixSum<<< grid, block >>>( deviceA, deviceB, deviceC );
    // データ転送: device -> host
    hipMemcpy( hostC, deviceC, matrixSize, hipMemcpyDeviceToHost );


    // 結果表示
    // puts("matrixA =");
    // showMatrix( hostA );
    // puts("matrixB =");
    // showMatrix( hostB );
    // puts("matrixC =");
    // showMatrix( hostC );


    // 時間計測終了
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    // 計測結果表示
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf("elapsed time: %f ms\n", elapsedTime);

    hipEventDestroy( start );
    hipEventDestroy( stop );


    // デバイス側のメモリ領域解放
    hipFree( deviceA );
    hipFree( deviceB );
    hipFree( deviceC );

    // ホスト側のメモリ領域解放
    free( hostA );
    free( hostB );
    free( hostC );

    return 0;
}


void showMatrix(int *matrix)
{
    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            printf("%5d ", matrix[i * SIZE + j]);
        }
        puts("");
    }
}


__global__ 
void matrixSum(int *matrixA, int *matrixB, int *matrixC)
{
    unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    matrixC[(y * SIZE) + x] 
        = matrixA[(y * SIZE) + x] + matrixB[(y * SIZE) + x];
}
